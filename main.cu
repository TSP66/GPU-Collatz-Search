#include "hip/hip_runtime.h"
#include <iostream>

#define BUFFER_SIZE 128 //Corresponds to a max number of 2^128
#define JOBS_THREAD 1024
#define BLOCK_SIZE 32
#define thread_x threadIdx.x
#define block_x blockIdx.x

__device__ void intToBin(bool * array, long long int value){
    int count = 0;
    for (int i = value; i > 0; i >>= 1){
        array[count++] = i%2;
    }
}

__device__ void plusOne(bool * array){
    bool carry = true;
    int pos = 0;
    while(carry){
        bool new_carry = array[pos] && carry;
        array[pos] = array[pos] || carry;
        carry = new_carry;
        pos++;
    }
}

__device__ void addition(bool * array, const bool * array2, bool carry){
    for (int i = BUFFER_SIZE; i > 0; i--){
        int sum = array[i]+array2[i]+carry;
        carry = (sum > 1);
        array[i] = (sum % 2);
    }
}

__device__ void divideTwo(bool * array){
    for (int i = BUFFER_SIZE - 1; i > 0; i--) {
        array[i] = array[i - 1];
    }
}

__device__ void timesThreeOne(bool * array){
    bool copy[BUFFER_SIZE];
    for (int i = 0; i < BUFFER_SIZE; i++)
        copy[i] = array[i];
    addition(array,copy,false);
    addition(array,copy,true);
}

__device__ void timesThreeOneDivideTwo(bool * array){
    timesThreeOne(array);
    divideTwo(array);
}

__device__ void isOne(bool * array){
    if (!array[BUFFER_SIZE-1]) return false;
    for (int i = 0; i < BUFFER_SIZE-1; i++){
        if (array[i]){
            return false;
        }
    }
}


__device__ void search(const bool * start){

    int jobs = JOBS_THREAD-1;
    long long int count = 0;
    bool current_number[BUFFER_SIZE];
    long long int trips[BUFFER_SIZE];

    bool array[BUFFER_SIZE];

    for (int i = 0; i < BUFFER_SIZE; i++){
        current_number[i] = start[i];
        array[i] = start[i]
    }

    while(jobs > 1){

        if (array[BUFFER_SIZE-1]){
            timesThreeOneDivideTwo(array);
        } else {
            divideTwo(array);
        }
        if (isOne(array)){
            trips[jobs] = count;
            plusOne(current_number);
            for (int i = 0; i < BUFFER_SIZE; i++){
                array[i] = current_number[i];
            }
            jobs--;
        }
        count++;
    }
}

__global__ void colatz_search(const bool * start){

}

int main(){
    return 0;
}